#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
CUDA C extension for Python
Provides functionality for forward projection in PET image
reconstruction.

author: Pawel Markiewicz
Copyrights: 2018
------------------------------------------------------------------------*/
#include "auxmath.h"
#include "prjf.h"
#include "tprj.h"

__constant__ float2 c_li2rng[NLI2R];
__constant__ short2 c_li2sn[NLI2R];
__constant__ char c_li2nos[NLI2R];

//===============================================================
// copy the smaller axially image to the one with full axial extension
__global__ void imExpand(float *im, float *imr, int vz0, int nvz) {
  int iz = vz0 + threadIdx.x;
  int iy = SZ_IMZ * threadIdx.y + SZ_IMZ * blockDim.y * blockIdx.x;
  if (iy < SZ_IMY * SZ_IMZ) {
    int idx = SZ_IMZ * SZ_IMY * blockIdx.y + iy + iz;
    int idxr = threadIdx.x + (nvz * threadIdx.y + nvz * blockDim.y * blockIdx.x) +
               nvz * SZ_IMY * blockIdx.y;
    // copy to the axially smaller image
    im[idx] = imr[idxr];
  }
}
//===============================================================

//**************** DIRECT ***********************************
__global__ void fprj_drct(float *sino, const float *im, const float *tt, const unsigned char *tv,
                          const int *subs, const short snno, const char span, const char att) {
  int ixt = subs[blockIdx.x]; // transaxial indx
  int ixz = threadIdx.x;      // axial (z)

  float z = c_li2rng[ixz].x + .5 * SZ_RING;
  int w = (floorf(.5 * SZ_IMZ + SZ_VOXZi * z));

  // if(ixz==33 && ixt==5301){
  //   printf("\n*** li2rng[ixz] = %f | li2sn[ixz] = %d, li2nos[ixz] = %d\n", li2rng[ixz],
  //   li2sn[ixz], li2nos[ixz]);
  // }

  //-------------------------------------------------
  /*** accumulation ***/
  // vector a (at) component signs
  int sgna0 = tv[N_TV * ixt] - 1;
  int sgna1 = tv[N_TV * ixt + 1] - 1;
  bool rbit = tv[N_TV * ixt + 2] & 0x01; // row bit

  int u = (int)tt[N_TT * ixt + 8];
  int v = (u >> UV_SHFT);
  int uv = SZ_IMZ * ((u & 0x000001ff) + SZ_IMX * v);

  // if((ixz==0) && (u>SZ_IMX || v>SZ_IMY)) printf("\n!!! u,v = %d,%d\n", u,v );

  // next voxel (skipping the first fractional one)
  uv += !rbit * sgna0 * SZ_IMZ;
  uv -= rbit * sgna1 * SZ_IMZ * SZ_IMX;

  float dtr = tt[N_TT * ixt + 2];
  float dtc = tt[N_TT * ixt + 3];

  float trc = tt[N_TT * ixt] + rbit * dtr;
  float tcc = tt[N_TT * ixt + 1] + dtc * !rbit;
  rbit = tv[N_TV * ixt + 3] & 0x01;

  float tn = trc * rbit + tcc * !rbit; // next t
  float tp = tt[N_TT * ixt + 5];       // previous t

  float lt, acc = 0;
  //-------------------------------------------------

  for (int k = 3; k < (int)tt[N_TT * ixt + 9]; k++) { //<<<< k=3, was k=2
    lt = tn - tp;
    acc += lt * im[w + uv];
    trc += dtr * rbit;
    tcc += dtc * !rbit;
    uv += !rbit * sgna0 * SZ_IMZ;
    uv -= rbit * sgna1 * SZ_IMZ * SZ_IMX;
    tp = tn;
    rbit = tv[N_TV * ixt + k + 1] & 0x01;
    tn = trc * rbit + tcc * !rbit;
  }

  if (att == 1) {
    if (span == 1)
      sino[c_li2sn[ixz].x + blockIdx.x * snno] = expf(-acc);
    else if (span == 11)
      atomicAdd(sino + c_li2sn[ixz].x + blockIdx.x * snno, expf(-acc) / (float)c_li2nos[ixz]);
  } else if (att == 0)
    atomicAdd(sino + c_li2sn[ixz].x + blockIdx.x * snno, acc);
}

//************** OBLIQUE **************************************************
__global__ void fprj_oblq(float *sino, const float *im, const float *tt, const unsigned char *tv,
                          const int *subs, const short snno, const char span, const char att,
                          const int zoff, const short nil2r_c) {
  int ixz = threadIdx.x + zoff; // axial (z)

  // if (ixz < NLI2R) {

  //> get the number of linear indices of direct and oblique sinograms
  if (ixz < nil2r_c) {

    int ixt = subs[blockIdx.x]; // transaxial index

    //-------------------------------------------------
    /*** accumulation ***/
    // vector a (at) component signs
    int sgna0 = tv[N_TV * ixt] - 1;
    int sgna1 = tv[N_TV * ixt + 1] - 1;
    bool rbit = tv[N_TV * ixt + 2] & 0x01; // row bit

    int u = (int)tt[N_TT * ixt + 8];
    int v = (u >> UV_SHFT);
    int uv = SZ_IMZ * ((u & 0x000001ff) + SZ_IMX * v);
    // next voxel (skipping the first fractional one)
    uv += !rbit * sgna0 * SZ_IMZ;
    uv -= rbit * sgna1 * SZ_IMZ * SZ_IMX;

    float dtr = tt[N_TT * ixt + 2];
    float dtc = tt[N_TT * ixt + 3];

    float trc = tt[N_TT * ixt] + rbit * dtr;
    float tcc = tt[N_TT * ixt + 1] + dtc * !rbit;
    rbit = tv[N_TV * ixt + 3] & 0x01;

    float tn = trc * rbit + tcc * !rbit; // next t
    float tp = tt[N_TT * ixt + 5];       // previous t
                                         //--------------------------------------------------

    //**** AXIAL *****
    float atn = tt[N_TT * ixt + 7];
    float az = c_li2rng[ixz].y - c_li2rng[ixz].x;
    float az_atn = az / atn;
    float s_az_atn = sqrtf(az_atn * az_atn + 1);
    int sgnaz;
    if (az >= 0)
      sgnaz = 1;
    else
      sgnaz = -1;

    float pz = c_li2rng[ixz].x + .5 * SZ_RING;
    float z = pz + az_atn * tp; // here was t1 = tt[N_TT*ixt+4]<<<<<<<<
    int w = (floorf(.5 * SZ_IMZ + SZ_VOXZi * z));
    float lz1 = (ceilf(.5 * SZ_IMZ + SZ_VOXZi * z)) * SZ_VOXZ -
                .5 * SZ_IMZ * SZ_VOXZ; // w is like in matlab by one greater

    z = c_li2rng[ixz].y + .5 * SZ_RING - az_atn * tp; // here was t1 = tt[N_TT*ixt+4]<<<<<<<<<
    int w_ = (floorf(.5 * SZ_IMZ + SZ_VOXZi * z));
    z = pz + az_atn * tt[N_TT * ixt + 6]; // t2
    float lz2 = (floorf(.5 * SZ_IMZ + SZ_VOXZi * z)) * SZ_VOXZ - .5 * SZ_IMZ * SZ_VOXZ;
    int nz = fabsf(lz2 - lz1) / SZ_VOXZ; // rintf
    float tz1 = (lz1 - pz) / az_atn;     // first ray interaction with a row
    float tz2 = (lz2 - pz) / az_atn;     // last ray interaction with a row
    float dtz = (tz2 - tz1) / nz;
    float tzc = tz1;
    //****************

    float fr, lt, acc = 0, acc_ = 0;
    for (int k = 3; k < tt[N_TT * ixt + 9];
         k++) { //<<< k=3 as 0 and 1 are for sign and 2 is skipped
      lt = tn - tp;
      if ((tn - tzc) > 0) {
        fr = (tzc - tp) / lt;
        acc += fr * lt * s_az_atn * im[w + uv];
        acc_ += fr * lt * s_az_atn * im[w_ + uv];
        w += sgnaz;
        w_ -= sgnaz;
        acc += (1 - fr) * lt * s_az_atn * im[w + uv];
        acc_ += (1 - fr) * lt * s_az_atn * im[w_ + uv];
        tzc += dtz;
      } else {
        acc += lt * s_az_atn * im[w + uv];
        acc_ += lt * s_az_atn * im[w_ + uv];
      }

      trc += dtr * rbit;
      tcc += dtc * !rbit;

      uv += !rbit * sgna0 * SZ_IMZ;
      uv -= rbit * sgna1 * SZ_IMZ * SZ_IMY;

      tp = tn;
      rbit = tv[N_TV * ixt + k + 1] & 0x01;
      tn = trc * rbit + tcc * !rbit;
    }

    // blockIdx.x is the transaxial bin index
    if (att == 1) {
      if (span == 1) {
        sino[c_li2sn[ixz].x + blockIdx.x * snno] = expf(-acc);
        sino[c_li2sn[ixz].y + blockIdx.x * snno] = expf(-acc_);
      } else if (span == 11) {
        atomicAdd(sino + c_li2sn[ixz].x + blockIdx.x * snno, expf(-acc) / (float)c_li2nos[ixz]);
        atomicAdd(sino + c_li2sn[ixz].y + blockIdx.x * snno, expf(-acc_) / (float)c_li2nos[ixz]);
      }
    } else if (att == 0) {
      atomicAdd(sino + c_li2sn[ixz].x + blockIdx.x * snno, acc);
      atomicAdd(sino + c_li2sn[ixz].y + blockIdx.x * snno, acc_);
    }
  }
}

//--------------------------------------------------------------------------------------------------
void gpu_fprj(float *d_sn, float *d_im, float *li2rng, short *li2sn, char *li2nos, short *s2c,
              int *aw2ali, float *crs, int *subs, int Nprj, int Naw, int N0crs, Cnst Cnt,
              char att) {
  int dev_id;
  hipGetDevice(&dev_id);
  if (Cnt.LOG <= LOGDEBUG) printf("i> using CUDA device #%d\n", dev_id);

  //--- TRANSAXIAL COMPONENT
  float4 *d_crs;
  HANDLE_ERROR(hipMalloc(&d_crs, N0crs * sizeof(float4)));
  HANDLE_ERROR(hipMemcpy(d_crs, crs, N0crs * sizeof(float4), hipMemcpyHostToDevice));

  short2 *d_s2c;
  HANDLE_ERROR(hipMalloc(&d_s2c, AW * sizeof(short2)));
  HANDLE_ERROR(hipMemcpy(d_s2c, s2c, AW * sizeof(short2), hipMemcpyHostToDevice));

  float *d_tt;
  HANDLE_ERROR(hipMalloc(&d_tt, N_TT * AW * sizeof(float)));

  unsigned char *d_tv;
  HANDLE_ERROR(hipMalloc(&d_tv, N_TV * AW * sizeof(unsigned char)));
  HANDLE_ERROR(hipMemset(d_tv, 0, N_TV * AW * sizeof(unsigned char)));

  // array of subset projection bins
  int *d_subs;
  HANDLE_ERROR(hipMalloc(&d_subs, Nprj * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(d_subs, subs, Nprj * sizeof(int), hipMemcpyHostToDevice));
  //---

  //-----------------------------------------------------------------
  // RINGS: either all or a subset of rings can be used (span-1 feature only)
  //-----------------------------------------------------------------
  // number of rings customised and the resulting size of LUTs and voxels
  short nrng_c, nil2r_c, vz0, vz1, nvz;
  // number of sinos
  short snno = -1;
  if (Cnt.SPN == 1) {
    // number of direct rings considered
    nrng_c = Cnt.RNG_END - Cnt.RNG_STRT;
    // number of "positive" michelogram elements used for projection (can be smaller than the
    // maximum)
    nil2r_c = (nrng_c + 1) * nrng_c / 2;
    snno = nrng_c * nrng_c;
    // correct for the max. ring difference in the full axial extent (don't use ring range (1,63)
    // as for this case no correction)
    if (nrng_c == NRINGS) {
      snno -= 12;
      nil2r_c -= 6;
    }
  } else if (Cnt.SPN == 11) {
    snno = NSINOS11;
    nrng_c = NRINGS;
    nil2r_c = NLI2R;
  }
  // voxels in axial direction
  vz0 = 2 * Cnt.RNG_STRT;
  vz1 = 2 * (Cnt.RNG_END - 1);
  nvz = 2 * nrng_c - 1;
  if (Cnt.LOG <= LOGDEBUG) {
    printf("i> detector rings range: [%d, %d) => number of  sinos: %d\n", Cnt.RNG_STRT,
           Cnt.RNG_END, snno);
    printf("   corresponding voxels: [%d, %d] => number of voxels: %d\n", vz0, vz1, nvz);
  }

  //-----------------------------------------------------------------

  //--- FULLY 3D
  HANDLE_ERROR(hipMemset(d_sn, 0, Nprj * snno * sizeof(float)));

  // when rings are reduced expand the image to account for whole axial FOV
  if (nvz < SZ_IMZ) {
    float *d_imr = d_im; // save old pointer to reduced image input
    // reallocate full size
    HANDLE_ERROR(hipMalloc(&d_im, SZ_IMX * SZ_IMY * SZ_IMZ * sizeof(float)));
    // put zeros in the gaps of unused voxels
    HANDLE_ERROR(hipMemset(d_im, 0, SZ_IMX * SZ_IMY * SZ_IMZ * sizeof(float)));
    int nar = NIPET_CU_THREADS / nvz;
    dim3 THRD(nvz, nar, 1);
    dim3 BLCK((SZ_IMY + nar - 1) / nar, SZ_IMX, 1);
    imExpand<<<BLCK, THRD>>>(d_im, d_imr, vz0, nvz);
    HANDLE_ERROR(hipGetLastError());
  }

  // float *d_li2rng;  HANDLE_ERROR( hipMalloc(&d_li2rng, N0li*N1li*sizeof(float)) );
  // HANDLE_ERROR( hipMemcpy( d_li2rng, li2rng, N0li*N1li*sizeof(float), hipMemcpyHostToDevice)
  // );

  // int *d_li2sn;  HANDLE_ERROR(hipMalloc(&d_li2sn, N0li*N1li*sizeof(int)) );
  // HANDLE_ERROR( hipMemcpy( d_li2sn, li2sn, N0li*N1li*sizeof(int), hipMemcpyHostToDevice) );

  // int *d_li2nos;  HANDLE_ERROR( hipMalloc(&d_li2nos, N1li*sizeof(int)) );
  // HANDLE_ERROR( hipMemcpy( d_li2nos, li2nos, N1li*sizeof(int), hipMemcpyHostToDevice) );

  hipMemcpyToSymbol(HIP_SYMBOL(c_li2rng), li2rng, nil2r_c * sizeof(float2));
  hipMemcpyToSymbol(HIP_SYMBOL(c_li2sn), li2sn, nil2r_c * sizeof(short2));
  hipMemcpyToSymbol(HIP_SYMBOL(c_li2nos), li2nos, nil2r_c * sizeof(char));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  if (Cnt.LOG <= LOGDEBUG) printf("i> calculating sinograms via forward projection...");

  //------------DO TRANSAXIAL CALCULATIONS---------------------------------
  gpu_siddon_tx(d_crs, d_s2c, d_tt, d_tv);
  //-----------------------------------------------------------------------

  //============================================================================
  fprj_drct<<<Nprj, nrng_c>>>(d_sn, d_im, d_tt, d_tv, d_subs, snno, Cnt.SPN, att);
  HANDLE_ERROR(hipGetLastError());
  //============================================================================

  int zoff = nrng_c;
  //> number of oblique sinograms
  int Noblq = (nrng_c - 1) * nrng_c / 2;
  int Nz = ((Noblq + 127) / 128) * 128;

  //============================================================================
  fprj_oblq<<<Nprj, Nz / 2>>>(d_sn, d_im, d_tt, d_tv, d_subs, snno, Cnt.SPN, att, zoff, nil2r_c);
  HANDLE_ERROR(hipGetLastError());

  zoff += Nz / 2;
  fprj_oblq<<<Nprj, Nz / 2>>>(d_sn, d_im, d_tt, d_tv, d_subs, snno, Cnt.SPN, att, zoff, nil2r_c);
  HANDLE_ERROR(hipGetLastError());
  //============================================================================

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  // hipDeviceSynchronize();
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  if (Cnt.LOG <= LOGDEBUG) printf("DONE in %fs.\n", 0.001 * elapsedTime);

  if (nvz < SZ_IMZ) HANDLE_ERROR(hipFree(d_im));
  HANDLE_ERROR(hipFree(d_tt));
  HANDLE_ERROR(hipFree(d_tv));
  HANDLE_ERROR(hipFree(d_subs));
  HANDLE_ERROR(hipFree(d_crs));
  HANDLE_ERROR(hipFree(d_s2c));
}

//=======================================================================
void rec_fprj(float *d_sino, float *d_img, int *d_sub, int Nprj,

              float *d_tt, unsigned char *d_tv,

              float *li2rng, short *li2sn, char *li2nos,

              Cnst Cnt)

{

  int dev_id;
  hipGetDevice(&dev_id);
  if (Cnt.LOG <= LOGDEBUG) printf("i> using CUDA device #%d\n", dev_id);

  // get the axial LUTs in constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(c_li2rng), li2rng, NLI2R * sizeof(float2));
  hipMemcpyToSymbol(HIP_SYMBOL(c_li2sn), li2sn, NLI2R * sizeof(short2));
  hipMemcpyToSymbol(HIP_SYMBOL(c_li2nos), li2nos, NLI2R * sizeof(char));

  // number of sinos
  short snno = -1;
  if (Cnt.SPN == 1)
    snno = NSINOS;
  else if (Cnt.SPN == 11)
    snno = NSINOS11;

  //> number of oblique sinograms
  int Noblq = (NRINGS * (NRINGS - 1) - 12) / 2;
  //> number of threads (in the axial direction)
  int Nz = ((Noblq + 127) / 128) * 128;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  if (Cnt.LOG <= LOGDEBUG) printf("i> subset forward projection (Nprj=%d)... ", Nprj);

  //============================================================================
  fprj_drct<<<Nprj, NRINGS>>>(d_sino, d_img, d_tt, d_tv, d_sub, snno, Cnt.SPN, 0);
  HANDLE_ERROR(hipGetLastError());
  //============================================================================

  int zoff = NRINGS;
  //============================================================================
  fprj_oblq<<<Nprj, Nz / 2>>>(d_sino, d_img, d_tt, d_tv, d_sub, snno, Cnt.SPN, 0, zoff, NLI2R);
  HANDLE_ERROR(hipGetLastError());
  //============================================================================

  zoff += Nz / 2;
  //============================================================================
  fprj_oblq<<<Nprj, Nz / 2>>>(d_sino, d_img, d_tt, d_tv, d_sub, snno, Cnt.SPN, 0, zoff, NLI2R);
  HANDLE_ERROR(hipGetLastError());
  //============================================================================

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  if (Cnt.LOG <= LOGDEBUG) printf("DONE in %fs.\n", 0.001 * elapsedTime);

  hipDeviceSynchronize();

  return;
}
