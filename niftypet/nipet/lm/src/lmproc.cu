/*----------------------------------------------------------------------
CUDA C extension for Python
Provides functionality for list-mode data processing including
histogramming on the GPU.

author: Pawel Markiewicz
Copyrights: 2020
----------------------------------------------------------------------*/

#include "lmproc.h"

void lmproc(
    hstout dicout,
    char *flm,
    int tstart,
    int tstop,
    int *c2sF,
    axialLUT axLUT,
    Cnst Cnt)

/*
Prepare for processing the list mode data and send it for GPU
execution.
*/
{

  // list mode data file (binary)
  if (Cnt.LOG <= LOGINFO) printf("i> the list-mode file: %s\n", flm);

    //------------ file and path names
#ifdef WIN32
  char *lmdir = strdup(flm);
#else
  char *lmdir = strdupa(flm);
#endif

  char *base = strrchr(lmdir, '/');
  lmdir[base - lmdir] = '\0';
  //------------



  //****** get LM info ******
  // uses global variable lmprop (see lmaux.cu)
  getLMinfo(flm, Cnt);
  //******


  //--- sino views for motion visualisation
  // already copy variables to output (number of time tags)
  dicout.nitag = lmprop.nitag;
  if (lmprop.nitag > MXNITAG)
    dicout.sne = MXNITAG / (1 << VTIME) * SEG0 * NSBINS;
  else
    dicout.sne = (lmprop.nitag + (1 << VTIME) - 1) / (1 << VTIME) * SEG0 * NSBINS;
  //---

  //--- sinograms in span-1 or span-11 or ssrb
  unsigned int tot_bins;

  if (Cnt.SPN == 1) {
    tot_bins = TOT_BINS_S1;
  } else if (Cnt.SPN == 11) {
    tot_bins = TOT_BINS;
  } else if (Cnt.SPN == 0) {
    tot_bins = SEG0 * NSBINANG;
  }
  //---

  //--- start and stop time
  //> if start and end times are equal (e.g., both '0')
  if (tstart == tstop) {
    tstart = 0;
    tstop = lmprop.nitag;
  }

  //> modify it in the properties variable
  lmprop.tstart = tstart;
  lmprop.tstop = tstop;
  
  //> bytes per LM event
  lmprop.bpe = Cnt.BPE;
  
  //> list mode data offset, start of events
  lmprop.lmoff = Cnt.LMOFF;
  //---


  if (Cnt.LOG <= LOGDEBUG) printf("i> LM offset in bytes: %d\n", lmprop.lmoff);
  if (Cnt.LOG <= LOGDEBUG) printf("i> bytes per LM event: %d\n", lmprop.bpe);
  if (Cnt.LOG <= LOGINFO) printf("i> frame start time: %d\n", tstart);
  if (Cnt.LOG <= LOGINFO) printf("i> frame stop  time: %d\n", tstop);
  //---


  if (Cnt.LOG <= LOGDEBUG)
    printf("ic> setting up all CUDA arrays...");

  //--- prompt & delayed reports
  unsigned int *d_rdlyd;
  unsigned int *d_rprmt;
  HANDLE_ERROR(hipMalloc(&d_rdlyd, lmprop.nitag * sizeof(unsigned int)));
  HANDLE_ERROR(hipMalloc(&d_rprmt, lmprop.nitag * sizeof(unsigned int)));

  HANDLE_ERROR(hipMemset(d_rdlyd, 0, lmprop.nitag * sizeof(unsigned int)));
  HANDLE_ERROR(hipMemset(d_rprmt, 0, lmprop.nitag * sizeof(unsigned int)));
  //---

  //--- for motion detection (centre of Mass)
  mMass d_mass;
  hipMalloc(&d_mass.zR, lmprop.nitag * sizeof(int));
  hipMalloc(&d_mass.zM, lmprop.nitag * sizeof(int));
  hipMemset(d_mass.zR, 0, lmprop.nitag * sizeof(int));
  hipMemset(d_mass.zM, 0, lmprop.nitag * sizeof(int));
  //---

  // motion visualisation video projections
  unsigned int *d_snview;
  if (lmprop.nitag > MXNITAG) {
    // reduce the sino views to only the first 2 hours
    hipMalloc(&d_snview, dicout.sne * sizeof(unsigned int));
    hipMemset(d_snview, 0, dicout.sne * sizeof(unsigned int));
  } else {
    hipMalloc(&d_snview, dicout.sne * sizeof(unsigned int));
    hipMemset(d_snview, 0, dicout.sne * sizeof(unsigned int));
  }
  //---

  //--- fansums for randoms estimation
  unsigned int *d_fansums;
  hipMalloc(&d_fansums, NRINGS * nCRS * sizeof(unsigned int));
  hipMemset(d_fansums, 0, NRINGS * nCRS * sizeof(unsigned int));
  //---

  //--- singles (buckets)
  // double the size as additionally saving the number of single
  // reports per second (there may be two singles' readings...)
  unsigned int *d_bucks;
  hipMalloc(&d_bucks, 2 * NBUCKTS * lmprop.nitag * sizeof(unsigned int));
  hipMemset(d_bucks, 0, 2 * NBUCKTS * lmprop.nitag * sizeof(unsigned int));
  //---

  //--- SSRB sino
  unsigned int *d_ssrb;
  HANDLE_ERROR(hipMalloc(&d_ssrb, SEG0 * NSBINANG * sizeof(unsigned int)));
  HANDLE_ERROR(hipMemset(d_ssrb, 0, SEG0 * NSBINANG * sizeof(unsigned int)));
  //---

  // prompt and delayed sinograms
  unsigned int *d_psino; //, *d_dsino;

  // prompt and compressed delayeds in one sinogram (two unsigned shorts)
  HANDLE_ERROR(hipMalloc(&d_psino, tot_bins * sizeof(unsigned int)));
  HANDLE_ERROR(hipMemset(d_psino, 0, tot_bins * sizeof(unsigned int)));

  //> important look-up table (LUT) for histogramming
  int *d_c2sF;
  HANDLE_ERROR(hipMalloc((void **)&d_c2sF, Cnt.NCRS*Cnt.NCRS * sizeof(int)));
  HANDLE_ERROR(hipMemcpy(d_c2sF, c2sF, Cnt.NCRS*Cnt.NCRS * sizeof(int), hipMemcpyHostToDevice));


  if (Cnt.LOG <= LOGDEBUG)
    printf("DONE\n");


  //======= get only the chunks which have the time frame data
  modifyLMinfo(tstart, tstop, Cnt);
  lmprop.span = Cnt.SPN;
  //===========

  //<><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><><>

  //**************************************************************************************
  gpu_hst(
    d_psino,
    d_ssrb,
    d_rdlyd,
    d_rprmt,
    d_fansums,
    d_bucks,
    d_mass,
    d_snview,
    tstart, tstop,
    d_c2sF, axLUT, Cnt);
  //**************************************************************************************
  hipDeviceSynchronize();

  dicout.tot = tot_bins;

  //---SSRB
  HANDLE_ERROR(hipMemcpy(dicout.ssr, d_ssrb, SEG0 * NSBINANG * sizeof(unsigned int),
                          hipMemcpyDeviceToHost));
  unsigned long long psum_ssrb = 0;
  for (int i = 0; i < SEG0 * NSBINANG; i++) { psum_ssrb += dicout.ssr[i]; }
  //---

  // //> copy to host the compressed prompt and delayed sinograms
  // unsigned int *sino = (unsigned int *)malloc(tot_bins * sizeof(unsigned int));
  // HANDLE_ERROR(hipMemcpy(sino, d_psino, tot_bins * sizeof(unsigned int), hipMemcpyDeviceToHost));

  // unsigned int mxbin = 0;
  // dicout.psm = 0;
  // dicout.dsm = 0;
  // for (int i = 0; i < tot_bins; i++) {
  //   dicout.psn[i] = sino[i] & 0x0000FFFF;
  //   dicout.dsn[i] = sino[i] >> 16;
  //   dicout.psm += dicout.psn[i];
  //   dicout.dsm += dicout.dsn[i];
  //   if (mxbin < dicout.psn[i]) mxbin = dicout.psn[i];
  // }

  // //--- output data to Python
  // // projection views
  // HANDLE_ERROR(
  //     hipMemcpy(dicout.snv, d_snview, dicout.sne * sizeof(unsigned int), hipMemcpyDeviceToHost));

  // // head curves
  // HANDLE_ERROR(hipMemcpy(dicout.hcd, d_rdlyd, lmprop.nitag * sizeof(unsigned int),
  //                         hipMemcpyDeviceToHost));
  // HANDLE_ERROR(hipMemcpy(dicout.hcp, d_rprmt, lmprop.nitag * sizeof(unsigned int),
  //                         hipMemcpyDeviceToHost));

  // // //mass centre
  // int *zR = (int *)malloc(lmprop.nitag * sizeof(int));
  // int *zM = (int *)malloc(lmprop.nitag * sizeof(int));
  // hipMemcpy(zR, d_mass.zR, lmprop.nitag * sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(zM, d_mass.zM, lmprop.nitag * sizeof(int), hipMemcpyDeviceToHost);

  // //> calculate the centre of mass while also the sum of head-curve prompts and delayeds
  // unsigned long long sphc = 0, sdhc = 0;
  // for (int i = 0; i < lmprop.nitag; i++) {
  //   dicout.mss[i] = zR[i] / (float)zM[i];
  //   sphc += dicout.hcp[i];
  //   sdhc += dicout.hcd[i];
  // }

  // if (Cnt.LOG <= LOGINFO)
  //   printf("\nic> total prompt single slice rebinned sinogram:  P = %llu\n", psum_ssrb);
  // if (Cnt.LOG <= LOGINFO)
  //   printf("\nic> total prompt and delayeds sinogram   events:  P = %llu, D = %llu\n", dicout.psm,
  //          dicout.dsm);
  // if (Cnt.LOG <= LOGINFO)
  //   printf("\nic> total prompt and delayeds head-curve events:  P = %llu, D = %llu\n", sphc, sdhc);
  // if (Cnt.LOG <= LOGINFO) printf("\nic> maximum prompt sino value:  %u \n", mxbin);

  // //-fansums and bucket singles
  // HANDLE_ERROR(hipMemcpy(dicout.fan, d_fansums, NRINGS * nCRS * sizeof(unsigned int),
  //                         hipMemcpyDeviceToHost));
  // HANDLE_ERROR(hipMemcpy(dicout.bck, d_bucks, 2 * NBUCKTS * lmprop.nitag * sizeof(unsigned int),
  //                         hipMemcpyDeviceToHost));

  // /* Clean up. */
  // free(zR);
  // free(zM);

  free(lmprop.atag);
  free(lmprop.btag);
  free(lmprop.ele4chnk);
  free(lmprop.ele4thrd);

  hipFree(d_psino);
  hipFree(d_ssrb);
  hipFree(d_rdlyd);
  hipFree(d_rprmt);
  hipFree(d_snview);
  hipFree(d_bucks);
  hipFree(d_fansums);
  hipFree(d_mass.zR);
  hipFree(d_mass.zM);
  hipFree(d_c2sF);

  return;
}
